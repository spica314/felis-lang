
#include <hip/hip_runtime.h>
extern "C" {
    __global__ void f(long *rs, long *gs, long *bs) {
        long thread_id = blockIdx.x * blockDim.x + threadIdx.x;

        long h = 256;
        long w = 256;
        long x = thread_id % w;
        long y = thread_id / w;

        float r = (float)x / (float)w;
        float g = (float)y / (float)h;
        float b = 0.2f;

        long ir = (long)(r * 255.99f);
        long ig = (long)(g * 255.99f);
        long ib = (long)(b * 255.99f);

        rs[thread_id] = ir;
        gs[thread_id] = ig;
        bs[thread_id] = ib;
    }
}
